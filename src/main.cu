#include <cutlass/gemm/device/gemm.h>
#include <iostream>

// Define matrix dimensions
constexpr int M = 128, N = 128, K = 128;

int main() {
    // Define CUTLASS GEMM
    using Gemm = cutlass::gemm::device::Gemm<
        float, cutlass::layout::RowMajor,
        float, cutlass::layout::RowMajor,
        float, cutlass::layout::RowMajor>;

    // Allocate and initialize device memory
    float *A, *B, *C;
    hipMalloc(&A, M * K * sizeof(float));
    hipMalloc(&B, K * N * sizeof(float));
    hipMalloc(&C, M * N * sizeof(float));

    // Run CUTLASS GEMM
    Gemm gemm_op;
    cutlass::gemm::GemmCoord problem_size(M, N, K);
    typename Gemm::Arguments args{problem_size, {A, K}, {B, N}, {C, N}, {C, N}, {1.0f, 0.0f}};
    gemm_op(args);

    hipDeviceSynchronize();
    std::cout << "CUTLASS GEMM completed.\n";

    hipFree(A); hipFree(B); hipFree(C);
    return 0;
}
